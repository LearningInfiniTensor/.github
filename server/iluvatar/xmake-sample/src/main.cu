﻿
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void hello() {
    printf("Hello, this is thread[%d.%d].\n", blockIdx.x, threadIdx.x);
}

int main() {
    hello<<<2, 4>>>();
    return 0;
}
